#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

// includes thrust
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/reduce.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>

using namespace thrust::placeholders;

////////////////////////////////////////////////////////////////////////////////
// Inline functions
////////////////////////////////////////////////////////////////////////////////

inline __device__ float calculateDistanceSquared(
    float x1, float y1, float z1,
    float x2, float y2, float z2) {
    return (x1 - x2)*(x1 - x2) +
           (y1 - y2)*(y1 - y2) +
           (z1 - z2)*(z1 - z2);
}

inline float random_float(){ 
    return (float)rand()/(float)RAND_MAX; 
}

////////////////////////////////////////////////////////////////////////////////
// Kernels
////////////////////////////////////////////////////////////////////////////////

__global__ void
calculateDistances(float *points_x, float* points_y, float* points_z,
        float *centroid_x, float* centroid_y, float* centroid_z,
        int *closest, int k, int n)
{
    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < n){
        unsigned index_min;
        float min_distance = 10e9;
        for(int i = 0; i < k; i++){
            float distance = calculateDistanceSquared(
                    points_x[tid], points_y[tid], points_z[tid],
                    centroid_x[i], centroid_y[i], centroid_z[i]);
            if(min_distance > distance){
                index_min = i;
                min_distance = distance;
            }
        }
        closest[tid] = index_min;
    }
}

__global__ void reduce(
        float *points_x, float* points_y, float* points_z,
        float *centroid_x, float* centroid_y, float* centroid_z,
        int *closest, int *sums, int n) {
    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < n){
        int centroid_num = closest[tid];
        atomicAdd(&centroid_x[centroid_num], points_x[tid]);
        atomicAdd(&centroid_y[centroid_num], points_y[tid]);
        atomicAdd(&centroid_z[centroid_num], points_z[tid]);
        atomicAdd(&sums[centroid_num], 1);
    }
}

__global__ void
calculateMean(float *centroid_x, float* centroid_y, float* centroid_z, int *sums, int k)
{
    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < k){
        int sum = sums[tid];
        if(sum != 0){
            centroid_x[tid] /= sums[tid];
            centroid_y[tid] /= sums[tid];
            centroid_z[tid] /= sums[tid];
        } else {
            centroid_x[tid] = 0.0f;
            centroid_y[tid] = 0.0f;
            centroid_z[tid] = 0.0f;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
// Function headers
////////////////////////////////////////////////////////////////////////////////

void randomize(thrust::host_vector<float>& point_x, thrust::host_vector<float>& point_y, thrust::host_vector<float>& point_z,
        thrust::host_vector<float>& centroid_x, thrust::host_vector<float>& centroid_y, thrust::host_vector<float>& centroid_z,
        int k, int n);

bool stop(thrust::host_vector<float>& h_centroid_x, thrust::host_vector<float>& h_centroid_y, thrust::host_vector<float>& h_centroid_z,
        thrust::device_vector<float>& d_centroid_x, thrust::device_vector<float>& d_centroid_y, thrust::device_vector<float>& d_centroid_z,
        int k, float epsilon);

void write(thrust::host_vector<float>& h_x, thrust::host_vector<float>& h_y, thrust::host_vector<float>& h_z, int n, const char* filename);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv){
    srand(0);
    //setup parameters
    int k = 8, n = 3200000;
    float epsilon = 0.0001f;
    //initialize host vectors
    thrust::host_vector<float> h_points_x(n), h_points_y(n), h_points_z(n);
    thrust::host_vector<float> h_centroids_x(k), h_centroids_y(k), h_centroids_z(k);
    //generate data
    randomize(h_points_x, h_points_y, h_points_z, h_centroids_x, h_centroids_y, h_centroids_z, k, n);
    //initialize device vectors, copy data from host vectors
    thrust::device_vector<float> d_points_x(h_points_x), d_points_y(h_points_y), d_points_z(h_points_z);
    thrust::device_vector<float> d_centroids_x = h_centroids_x, d_centroids_y = h_centroids_y, d_centroids_z = h_centroids_z;
    thrust::device_vector<int> d_closest(n);
    thrust::device_vector<int> d_sums(k);
    //start timers
    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // setup execution parameters
    dim3  grid(n / 256 + 1, 1, 1);
    dim3  threads(256, 1, 1);

    dim3  grid2(k / 1024 + 1, 1, 1);
    dim3  threads2(1024, 1, 1);

    int iter = 0;
    do {
        thrust::fill(d_closest.begin(), d_closest.end(), 0);

        //for each point in data set find closest centroid
        calculateDistances<<< grid, threads >>>(
                thrust::raw_pointer_cast(&d_points_x[0]),
                thrust::raw_pointer_cast(&d_points_y[0]),
                thrust::raw_pointer_cast(&d_points_z[0]),
                thrust::raw_pointer_cast(&d_centroids_x[0]),
                thrust::raw_pointer_cast(&d_centroids_y[0]),
                thrust::raw_pointer_cast(&d_centroids_z[0]),
                thrust::raw_pointer_cast(&d_closest[0]),
                k, n);

        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");
        
        //clear old centroids data
        thrust::fill(d_centroids_x.begin(), d_centroids_x.end(), 0.0f);
        thrust::fill(d_centroids_y.begin(), d_centroids_y.end(), 0.0f);
        thrust::fill(d_centroids_z.begin(), d_centroids_z.end(), 0.0f);
        thrust::fill(d_sums.begin(), d_sums.end(), 0);

        //sum up for each centroid distance to point from point's perspective
        reduce<<< grid, threads >>>(
                thrust::raw_pointer_cast(&d_points_x[0]),
                thrust::raw_pointer_cast(&d_points_y[0]),
                thrust::raw_pointer_cast(&d_points_z[0]),
                thrust::raw_pointer_cast(&d_centroids_x[0]),
                thrust::raw_pointer_cast(&d_centroids_y[0]),
                thrust::raw_pointer_cast(&d_centroids_z[0]),
                thrust::raw_pointer_cast(&d_closest[0]),
                thrust::raw_pointer_cast(&d_sums[0]),
                n);
        
        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");

        //now calculate mean from the previously calculated sum it is a new centroid
        calculateMean<<< grid2, threads2 >>>(
                thrust::raw_pointer_cast(&d_centroids_x[0]),
                thrust::raw_pointer_cast(&d_centroids_y[0]),
                thrust::raw_pointer_cast(&d_centroids_z[0]),
                thrust::raw_pointer_cast(&d_sums[0]), k);

        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");
        //one iteration done
        iter = iter + 1;
    } while(
        //check if change is small compared to the last iteration
        !stop(h_centroids_x, h_centroids_y, h_centroids_z, 
            d_centroids_x, d_centroids_y, d_centroids_z,
            k, epsilon) || iter > 100);

    //stop timers and print summary
    sdkStopTimer(&timer);
    printf("Processing time: %f (ms), %d iterations\n", sdkGetTimerValue(&timer), iter);
    sdkDeleteTimer(&timer);
    
    //write output of the program to a file
    write(h_points_x, h_points_y, h_points_z, n, "points.txt");
    write(h_centroids_x, h_centroids_y, h_centroids_z, k, "centroids.txt");

    printf("Exiting...\n");
    exit(EXIT_SUCCESS);
}

//generate data
void randomize(thrust::host_vector<float>& point_x, thrust::host_vector<float>& point_y, thrust::host_vector<float>& point_z,
    thrust::host_vector<float>& centroid_x, thrust::host_vector<float>& centroid_y, thrust::host_vector<float>& centroid_z,
    int k, int n){
    for(int i = 0; i < k; i++){
        float x = random_float();
        float y = random_float();
        float z = random_float();
        centroid_x[i] = x;
        centroid_y[i] = y;
        centroid_z[i] = z;
    }
    for(int i = 0; i < n; i++){
        float x = random_float();
        float y = random_float();
        float z = random_float();
        point_x[i] = x;
        point_y[i] = y;
        point_z[i] = z;
    }
}

//check if alghoritm should stop, i.e. if norm of centroids vector is lesser
//than given epsilon
bool stop(thrust::host_vector<float>& h_centroid_x, thrust::host_vector<float>& h_centroid_y, thrust::host_vector<float>& h_centroid_z,
    thrust::device_vector<float>& d_centroid_x, thrust::device_vector<float>& d_centroid_y, thrust::device_vector<float>& d_centroid_z,
    int k, float epsilon){
    thrust::host_vector<float> 
        h_centroid_x_new(d_centroid_x), 
        h_centroid_y_new(d_centroid_y), 
        h_centroid_z_new(d_centroid_z);
    float norm = 0.0f;
    for(int i = 0; i < k; i++){
        norm += abs(h_centroid_x_new[i] - h_centroid_x[i]) + 
                abs(h_centroid_y_new[i] - h_centroid_y[i]) + 
                abs(h_centroid_z_new[i] - h_centroid_z[i]);
    }
    norm /= (k * 3);
    h_centroid_x = h_centroid_x_new;
    h_centroid_y = h_centroid_y_new;
    h_centroid_z = h_centroid_z_new;
    printf("norm: %f\n", norm);
    if(norm > epsilon) return false;
    else return true;
}

// writes vectors to a specified file
void write(thrust::host_vector<float>& h_x, thrust::host_vector<float>& h_y, thrust::host_vector<float>& h_z, int n, const char* filename){
    std::ofstream myfile;
    myfile.open(filename);
    for(int i = 0; i < n; i++){
        myfile << h_x[i] << " " << h_y[i] << " " << h_z[i] << " " << std::endl;
    }
    myfile.close();
}
